#include "hip/hip_runtime.h"
#include "cumm.cuh"

#define THREAD_DIM 16


size_t cdiv(size_t total, size_t size){
    return (total + size - 1) / size;
}

__global__ void multiply_kernel(double *matA, double *matB, double *matC, int m, int k, int n){
    __shared__ double sharedA[THREAD_DIM][THREAD_DIM];
    __shared__ double sharedB[THREAD_DIM][THREAD_DIM];

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    double res = 0.;
    for (int tile = 0; tile < k; tile += THREAD_DIM){
        int colA = tile + threadIdx.y;
        int rowB = tile + threadIdx.x;
        sharedA[threadIdx.x][threadIdx.y] = (colA < k && row < m)? matA[row * k + colA]: 0.;
        sharedB[threadIdx.x][threadIdx.y] = (rowB < k && col < n)? matB[rowB * n + col]: 0.;
        __syncthreads();
        for (int i = 0; i < THREAD_DIM; i++)
            res += sharedA[threadIdx.x][i] * sharedB[i][threadIdx.y];
        __syncthreads();
    }

    if (row < m && col < n)
        matC[row * n + col] = res;
}

Matrix multiply_cuda(const Matrix &A, const Matrix &B){
    if (A.ncol() != B.nrow())
        throw std::runtime_error("A.ncol() does not equal to B.nrow().");
        
    Matrix C(A.nrow(), B.ncol());

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, A.buffer_size());
    hipMalloc(&d_B, B.buffer_size());
    hipMalloc(&d_C, C.buffer_size());

    hipMemcpy(d_A, A.data_ptr(), A.buffer_size(), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data_ptr(), B.buffer_size(), hipMemcpyHostToDevice);

    size_t m = A.nrow(), k = A.ncol(), n = B.ncol();
    dim3 thread_dims(THREAD_DIM, THREAD_DIM);
    dim3 block_dims(cdiv(m, THREAD_DIM), cdiv(n, THREAD_DIM));
    multiply_kernel<<<block_dims, thread_dims>>>(d_A, d_B, d_C, m, k, n);

    hipMemcpy(C.data_ptr(), d_C, C.buffer_size(), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return C;
}